#include "common.h"

void checkCUDAErrorFn(const char *msg, const char *file, int line) {
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": line %d: %s: %s\n", line, msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
}


namespace AES {
    namespace Common {



    }
}
