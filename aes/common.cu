#include "hip/hip_runtime.h"
#include "common.h"

int BLOCKSIZE = 256;
bool USING_PARAMETER = false;
bool USING_SHAREDMEM = true;
bool SHAREDMEM_KEY = true;
bool SHAREDMEM_SBOX = true;
bool CONSTANTMEM = false;
bool QUIET = false;
int AES_SIZE = 256;//other options 192, 128
int AES_KEY_EXP_SIZE = 240;
int ABLOCKS_PER_THREAD = 1;

void checkCUDAErrorFn(const char *msg, const char *file, int line) {
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": line %d: %s: %s\n", line, msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
}

bool ingestCommandLineOptions(int aes_size, int blocksize,
		int blocksperthread,
		bool sharedmem_key,
		bool sharedmem_sbox,
		bool parameter,
		bool quiet,
		bool constant) {

	if (aes_size == 256 || aes_size == 192 || aes_size == 128) {
		AES_SIZE = aes_size;
	}
	else {
		printf("ERROR: invalid AES size argument, must be 256, 192, or 128\n");
		return false;
	}
	if (blocksize >= 32 && blocksize % 32 == 0){
	//if (ceil(log2(blocksize)) == floor(log2(blocksize))) {
		BLOCKSIZE = blocksize;
	}
	else {
		printf("ERROR: invalid block size, must be multiple of 32\n");
		return false;
	}
	if (blocksperthread < 1 || blocksperthread > 1024) {
		printf("ERROR: invalid blocksperthread argument, must be between 1 and 1024\n");
		return false;
	}
	ABLOCKS_PER_THREAD	= blocksperthread;
	SHAREDMEM_KEY		= sharedmem_key;
	SHAREDMEM_SBOX		= sharedmem_sbox;
	USING_SHAREDMEM		= SHAREDMEM_KEY | SHAREDMEM_SBOX;
	USING_PARAMETER		= parameter;
	CONSTANTMEM			= constant;
	QUIET				= quiet;

	switch (AES_SIZE) {
	case 256:
		AES_KEY_EXP_SIZE = 240;
		break;
	case 192:
		AES_KEY_EXP_SIZE = 208;
		break;
	case 128:
		AES_KEY_EXP_SIZE = 176;
		break;
	default:
		break;
	}

	if (!QUIET) {
		printf("block %d, key %d, shared %d, skey %d, sbox %d, parameter %d\n",
			BLOCKSIZE, AES_SIZE, USING_SHAREDMEM, SHAREDMEM_KEY, SHAREDMEM_SBOX, USING_PARAMETER);
	}

	return true;
}//ingestCommandLineOptions


namespace AES {
    namespace Common {
		uint64_t padData(uint8_t* data, uint64_t currentLength) {
			uint64_t newLen = ((currentLength / AES_BLOCKLEN) + 1) * AES_BLOCKLEN;
			uint8_t lenDiff = (uint8_t)(newLen - currentLength);
			for (uint8_t i = 0; i < lenDiff; i++) {
				data[currentLength + i] = lenDiff;
			}

			return newLen;
		}//padData

		uint64_t unpadData(uint8_t* data, uint64_t currentLength) {
#if ASSERTING
			assert(currentLength % AES_BLOCKLEN == 0);
#endif
			uint8_t padAmount = data[currentLength - 1];
#if ASSERTING
			assert(padAmount <= AES_BLOCKLEN);
			assert(padAmount > 0);

			for (uint8_t i = 0; i < padAmount; i++) {
				assert(data[currentLength - 1 - i] == padAmount);
			}
#endif

			uint64_t newLen = currentLength - padAmount;
			return newLen;
		}


    }
}
